#include "hip/hip_runtime.h"
#include "cuda-runtime-api.h"

__device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy) {
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

__global__ void decode_kernel(float *predict, int num_bboxes, int num_classes, float confidence_threshold,
                              float *invert_affine_matrix, float *parray, int max_objects, int NUM_BOX_ELEMENT) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) { return; }

    // pitem 是指向每一行结果的首地址
    float *pitem = predict + (5 + num_classes) * position;
    // 置信度，表示该预测框中是否有对象的置信度。
    float objectness = pitem[4];
    if (objectness < confidence_threshold) { return; }

    // 当前行的 label 指针，指向类别概率的起始位置。
    float *class_confidence = pitem + 5;
    // 获取 class_confidence 所指向的浮点数的值，将其赋值给 confidence，然后将 class_confidence 指针递增到下一个位置。
    float confidence = *class_confidence++; // *ptr 解引用指针;
    int label = 0;

    // 此处的 for 循环相当于 std::max_element 的作用；
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label = i;
        }
    }
    confidence *= objectness;
    if (objectness < confidence_threshold) { return; }

    // 使用原子操作 atomicAdd 递增并获取当前输出索引 output_index。
    /*
     * parray = [count, box1, box2, ……]
     * atomicAdd(parray, 1) 的操作相当于是 count += 1, 但是返回的是没有 +1 之前的 old_count;
     */
    int index = atomicAdd(parray, 1);
    if (index >= max_objects) { return; }

    // 获取 left, top, width, height 的值
    float cx = *pitem++;
    float cy = *pitem++;
    float width = *pitem++;
    float height = *pitem++;

    // xywh to xyxy
    float left = cx - width * 0.5;
    float top = cy - height * 0.5;
    float right = cx + width * 0.5;
    float bottom = cy + height * 0.5;

    /*
     * 仿射变换
     * affine_project(invert_affine_matrix, left,  top,    &left,  &top);
     * affine_project(invert_affine_matrix, right, bottom, &right, &bottom);
     */

    // 将转换好的数据进行存储；
    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 用于nms的标志位，1 = keep, 0 = ignore
}

__device__ float box_iou(float aleft, float atop, float aright, float abottom,
                         float bleft, float btop, float bright, float bbottom) {
    float cleft = max(aleft, bleft);
    float ctop = max(atop, btop);
    float cright = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);

    if (c_area == 0.0f) { return 0.0f; }
    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

__global__ void fast_nms_kernel(float *bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT) {
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    // (int)*bboxes 解引用获取第一个计数器的值 count;
    int count = min((int)*bboxes, max_objects);
    if (position >= count) { return; }

    // left, top, right, bottom, confidence, class, keepflag
    float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) {
        float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5]) { continue; }

        if (pitem[4] >= pcurrent[4]) {
            // 过滤
            if (pitem[4] == pcurrent[4] && i < position) { continue; }

            float iou = box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                                pitem[0], pitem[1], pitem[2], pitem[3]);

            if (iou > threshold) {
                pcurrent[6] = 0;
                return;
            }
        }
    }
}

void decode_kernel_invoker(
    float *predict, int num_bboxes, int num_classes, float confidence_threshold,
    float nms_threshold, float *invert_affine_matrix, float *parray, int max_objects,
    int NUM_BOX_ELEMENT, hipStream_t stream) {
    auto block = num_bboxes > 512 ? 512 : num_bboxes;
    auto grid = (num_bboxes + block - 1) / block;

    // [left, top, width, height, confidence,label1, label2,……] ---> [left, top, width, height, confidence,label,nms_symbol]
    decode_kernel<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, confidence_threshold,
        invert_affine_matrix, parray, max_objects, NUM_BOX_ELEMENT);

    block = max_objects > 512 ? 512 : max_objects;
    grid = (max_objects + block - 1) / block;
    fast_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold, NUM_BOX_ELEMENT);
}